#include <iostream>
#include <vector>
#include <cstdio> 
#include <hip/hip_runtime.h>

__global__ void PrintThreadIdx (int* PrintId) {
    int Idx = blockIdx.x * blockDim.x + threadIdx.x;
    PrintId[Idx] = Idx;}

__global__ void EvenOddKernel (int* output, int N) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   if (tid < N) {
    if (tid % 2 == 0) {
        output[tid] = 1;
    } else{
        output[tid] = 0;
    }
   }
}

int main() {

      int N = 987;

    int* h_output = new int[N];
    int* d_output;
    hipMalloc((void**)&d_output, N * sizeof(int));

    int numThreadsPerBlock = 16;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;


    PrintThreadIdx <<<numBlocks, numThreadsPerBlock>>> (d_output);
    hipMemcpy(h_output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Thread IDs: " << std::endl;

    for (int i = 0; i < N; i++) {
        std::cout << h_output[i] << std::endl;
    }

    std::cout << std::endl;

    hipMemcpy(d_output, h_output, N * sizeof(int), hipMemcpyHostToDevice);
    
    EvenOddKernel<<<numBlocks, numThreadsPerBlock>>> (d_output, N);
    hipMemcpy(h_output, d_output, N * sizeof(int), hipMemcpyDeviceToHost);

    std::cout << "Even/Odd Markers: " << std::endl;
   for (int i = 0; i < N; i++) {
        std::cout << h_output[i] << std::endl;
    }

    std::cout << std::endl;

delete[] h_output;
hipFree(d_output);

return 0;
}
